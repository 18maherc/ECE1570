#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256

extern double size;
//
//  benchmarking program
//

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if (r2 > cutoff * cutoff)
        return;
    // r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    //
    //  very simple short-range repulsive force
    //
    double coef = (1 - cutoff / r) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;
}

__global__ void compute_forces_gpu(particle_t *particles, int n, particle_t *bin_grid, int *bin_grid_count)
{
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= n)
        return;

    particles[tid].ax = particles[tid].ay = 0;

    int sidelength = get_sidelength();

    // Compute what i and j values the particle is at
    double iIndex = 0;
    double jIndex = 0;

    double lowVal = 0;
    double highVal = cutoff;
    for (int j = 0; j < sidelength; j++)
    {
        if (particles[tid].x >= lowVal && particles[tid].x <= highVal)
        {
            iIndex = j;
        }
        if (particles[tid].y >= lowVal && particles[tid].y <= highVal)
        {
            jIndex = j;
        }

        lowVal += cutoff;
        highVal += cutoff;
    }

    // Top left bin
    if (
        (iIndex - 1) >= 0 &&
        (iIndex - 1) < sidelength &&
        (jIndex - 1) >= 0 &&
        (jIndex - 1) < sidelength)
    {
        int index2D = get2DIndex(iIndex - 1, jIndex - 1);
        int count = bin_grid_count[index2D];
        int index3D = get3DIndex(iIndex - 1, jIndex - 1, count);
        // Iterate through all other particles of that bin
        for (int j = 0; j < count; j++)
        {
            apply_force_gpu(
                particles[tid],
                bin_grid[index3D]);
        }
    }
    // Left bin
    if (
        (iIndex - 1) >= 0 &&
        (iIndex - 1) < sidelength &&
        (jIndex) >= 0 &&
        (jIndex) < sidelength)
    {
        int index2D = get2DIndex(iIndex - 1, jIndex);
        int count = bin_grid_count[index2D];
        int index3D = get3DIndex(iIndex - 1, jIndex, count);
        // Iterate through all other particles of that bin
        for (int j = 0; j < bin_grid.at(iIndex - 1).at(jIndex).size(); j++)
        {
            apply_force_gpu(
                particles[tid],
                bin_grid[index3D]);
        }
    }
    // Bot left bin
    if (
        (iIndex - 1) >= 0 &&
        (iIndex - 1) < sidelength &&
        (jIndex + 1) >= 0 &&
        (jIndex + 1) < sidelength)
    {
        int index2D = get2DIndex(iIndex - 1, jIndex + 1);
        int count = bin_grid_count[index2D];
        int index3D = get3DIndex(iIndex - 1, jIndex + 1, count);
        // Iterate through all other particles of that bin
        for (int j = 0; j < bin_grid.at(iIndex - 1).at(jIndex + 1).size(); j++)
        {
            apply_force_gpu(
                particles[tid],
                bin_grid[index3D]);
        }
    }
    // Top bin
    if (
        (iIndex) >= 0 &&
        (iIndex) < sidelength &&
        (jIndex - 1) >= 0 &&
        (jIndex - 1) < sidelength)
    {
        int index2D = get2DIndex(iIndex, jIndex - 1);
        int count = bin_grid_count[index2D];
        int index3D = get3DIndex(iIndex, jIndex - 1, count);
        // Iterate through all other particles of that bin
        for (int j = 0; j < bin_grid.at(iIndex).at(jIndex - 1).size(); j++)
        {
            apply_force_gpu(
                particles[tid],
                bin_grid[index3D]);
        }
    }
    // Top right bin
    if (
        (iIndex + 1) >= 0 &&
        (iIndex + 1) < sidelength &&
        (jIndex - 1) >= 0 &&
        (jIndex - 1) < sidelength)
    {
        int index2D = get2DIndex(iIndex + 1, jIndex - 1);
        int count = bin_grid_count[index2D];
        int index3D = get3DIndex(iIndex + 1, jIndex - 1, count);
        // Iterate through all other particles of that bin
        for (int j = 0; j < bin_grid.at(iIndex + 1).at(jIndex - 1).size(); j++)
        {
            apply_force_gpu(
                particles[tid],
                bin_grid[index3D]);
        }
    }
    // Right bin
    if (
        (iIndex + 1) >= 0 &&
        (iIndex + 1) < sidelength &&
        (jIndex) >= 0 &&
        (jIndex) < sidelength)
    {
        int index2D = get2DIndex(iIndex + 1, jIndex);
        int count = bin_grid_count[index2D];
        int index3D = get3DIndex(iIndex + 1, jIndex, count);
        // Iterate through all other particles of that bin
        for (int j = 0; j < bin_grid.at(iIndex + 1).at(jIndex).size(); j++)
        {
            apply_force_gpu(
                particles[tid],
                bin_grid[index3D]);
        }
    }
    // Bot right bin
    if (
        (iIndex + 1) >= 0 &&
        (iIndex + 1) < sidelength &&
        (jIndex + 1) >= 0 &&
        (jIndex + 1) < sidelength)
    {
        int index2D = get2DIndex(iIndex + 1, jIndex + 1);
        int count = bin_grid_count[index2D];
        int index3D = get3DIndex(iIndex + 1, jIndex + 1, count);
        // Iterate through all other particles of that bin
        for (int j = 0; j < bin_grid.at(iIndex + 1).at(jIndex + 1).size(); j++)
        {
            apply_force_gpu(
                particles[tid],
                bin_grid[index3D]);
        }
    }
    // Bot bin
    if (
        (iIndex) >= 0 &&
        (iIndex) < sidelength &&
        (jIndex + 1) >= 0 &&
        (jIndex + 1) < sidelength)
    {
        int index2D = get2DIndex(iIndex, jIndex + 1);
        int count = bin_grid_count[index2D];
        int index3D = get3DIndex(iIndex, jIndex + 1, count);
        // Iterate through all other particles of that bin
        for (int j = 0; j < bin_grid.at(iIndex).at(jIndex + 1).size(); j++)
        {
            apply_force_gpu(
                particles[tid],
                bin_grid[index3D]);
        }
    }
    // Middle bin
    if (
        (iIndex) >= 0 &&
        (iIndex) < sidelength &&
        (jIndex) >= 0 &&
        (jIndex) < sidelength)
    {
        int index2D = get2DIndex(iIndex, jIndex);
        int count = bin_grid_count[index2D];
        int index3D = get3DIndex(iIndex, jIndex, count);
        // Iterate through all other particles of that bin
        for (int j = 0; j < bin_grid.at(iIndex).at(jIndex).size(); j++)
        {
            apply_force_gpu(
                particles[tid],
                bin_grid[index3D]);
        }
    }

    // for(int j = 0 ; j < n ; j++) {
    //     apply_force_gpu(particles[tid], particles[j]);
    // }
}

__global__ void move_gpu(particle_t *particles, int n, double size)
{

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= n)
        return;

    particle_t *p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    //
    //  bounce from walls
    //
    while (p->x < 0 || p->x > size)
    {
        p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
        p->vx = -(p->vx);
    }
    while (p->y < 0 || p->y > size)
    {
        p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
        p->vy = -(p->vy);
    }
}

int main(int argc, char **argv)
{
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize();

    if (find_option(argc, argv, "-h") >= 0)
    {
        printf("Options:\n");
        printf("-h to see this help\n");
        printf("-n <int> to set the number of particles\n");
        printf("-o <filename> to specify the output file name\n");
        return 0;
    }

    int n = read_int(argc, argv, "-n", 1000);

    char *savename = read_string(argc, argv, "-o", NULL);

    FILE *fsave = savename ? fopen(savename, "w") : NULL;
    particle_t *particles = (particle_t *)malloc(n * sizeof(particle_t));

    // GPU particle data structure
    particle_t *d_particles;
    hipMalloc((void **)&d_particles, n * sizeof(particle_t));

    set_size(n);

    init_particles(n, particles);

    // Initialize our bins for O(n) shortcut checking
    int sidelength = get_sidelength();
    particle_t *d_bin_grid; // device bin grid
    hipMalloc(
        (void **) % d_bin_grid,
        sidelength * sidelength * n); // A bin may have up to n particles, so we'll set the inner size to n
    // This may be horribly inefficient
    int *d_bin_grid_count; // how many particles are in each bin
    hipMalloc(
        (void **) % d_bin_grid_count,
        sidelength * sidelength);

    hipDeviceSynchronize();
    double copy_time = read_timer();

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer() - copy_time;

    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer();

    for (int step = 0; step < NSTEPS; step++)
    {
        // Zero out all the matrices
        for (int i = 0; i < sidelength; i++)
        {
            for (int j = 0; j < sidelength; j++)
            {
                for (int k = 0; k < n; k++)
                {
                    int index3D = get3DIndex(i, j, k);
                    d_bin_grid[index3D].x = -1;
                    d_bin_grid[index3D].y = -1;
                }

                int index2D = get2DIndex(i, j);
                d_bin_grid_count[index2D] = 0;
            }
        }

        // Add all particles to the matrix in their proper grid locations
        for (int i = 0; i < n; i++)
        {
            // Compute what i and j values the particle is at
            double iIndex = 0;
            double jIndex = 0;

            double lowVal = 0;
            double highVal = cutoff;
            for (int j = 0; j < sidelength; j++)
            {
                if (d_particles[i].x >= lowVal && d_particles[i].x <= highVal)
                {
                    iIndex = j;
                }
                if (d_particles[i].y >= lowVal && d_particles[i].y <= highVal)
                {
                    jIndex = j;
                }

                lowVal += cutoff;
                highVal += cutoff;
            }

            // Place the particle in the grid
            int index2D = get2DIndex(iIndex, jIndex);
            int countForBin = d_bin_grid_count[index2D];
            int index3D = get3DIndex(iIndex, jIndex, countForBin);
            d_bin_grid[index3D] = d_particles[i]; // Place this particle
            d_bin_grid_count[index2D] += 1;       // Count this particle
        }

        //
        //  compute forces
        //

        int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
        compute_forces_gpu<<<blks, NUM_THREADS>>>(d_particles, n, d_bin_grid, d_bin_grid_count);

        //
        //  move particles
        //
        move_gpu<<<blks, NUM_THREADS>>>(d_particles, n, size);

        //
        //  save if necessary
        //
        if (fsave && (step % SAVEFREQ) == 0)
        {
            // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save(fsave, n, particles);
        }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer() - simulation_time;

    printf("CPU-GPU copy time = %g seconds\n", copy_time);
    printf("n = %d, simulation time = %g seconds\n", n, simulation_time);

    hipFree(d_bin_grid);
    hipFree(d_bin_grid_count);

    free(particles);
    hipFree(d_particles);
    if (fsave)
        fclose(fsave);

    return 0;
}
